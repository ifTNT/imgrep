#include "hip/hip_runtime.h"
extern "C" {
#include "match.h"
}
#include "cudautil.h"

extern "C" void match_general_gpu(match_result_t *result, flatmat_t *src,
                                  flatmat_t *tmpl, int blk_size, int thrd_size,
                                  match_thrd_t *region_calc) {
  // Initialize task parameter
  result->cnt = 0;
  result->height = src->height - tmpl->height + 1;
  result->width = src->width - tmpl->width + 1;
  uint thrd_cnt = blk_size * thrd_size;
  uint task_w = ceil((float)result->width / thrd_cnt);
  uint task_h = ceil((float)result->height / thrd_cnt);
  dim3 blk_size_2d(blk_size, blk_size);
  dim3 thrd_size_2d(thrd_size, thrd_size);
  match_work_region_t task_param{.off_x = 0, // Unused
                                 .off_y = 0, // Unused
                                 .width = task_w,
                                 .height = task_h,
                                 .bound_x = result->width,
                                 .bound_y = result->height};

  // Initialize device and host resource
  flatmat_t device_src, device_tmpl, device_similarly, device_result;
  flatmat_t host_similarly;

  flatmat_init(&host_similarly, result->width, result->height, 1);
  flatmat_init(&result->map, result->width, result->height, 1);
  flatmat_init_cuda(&device_similarly, result->width, result->height, 1);
  flatmat_init_cuda(&device_result, result->width, result->height, 1);
  flatmat_init_cuda(&device_src, src->width, src->height, src->layer);
  flatmat_init_cuda(&device_tmpl, tmpl->width, tmpl->height, tmpl->layer);

  // Copy data to device
  GPU_ERRCHK(
      hipMemcpy(device_src.el, src->el, src->size, hipMemcpyHostToDevice));
  GPU_ERRCHK(
      hipMemcpy(device_tmpl.el, tmpl->el, tmpl->size, hipMemcpyHostToDevice));

  // Calculate similarly map
  region_calc<<<blk_size_2d, thrd_size_2d>>>(device_similarly, device_src,
                                             device_tmpl, task_param);
  // Copy back the similarly map
  GPU_ERRCHK(hipMemcpy(host_similarly.el, device_similarly.el,
                        device_similarly.size, hipMemcpyDeviceToHost));

  GPU_ERRCHK(hipPeekAtLastError());
  GPU_ERRCHK(hipDeviceSynchronize());
  float max = -1.0E9;
  for (int y = 0; y < host_similarly.height; y++) {
    for (int x = 0; x < host_similarly.width; x++) {
      if (*FLATMAT_AT(&host_similarly, x, y, 0) > max)
        max = *FLATMAT_AT(&host_similarly, x, y, 0);
    }
  }

  // Draw the result map
  int tmp;
  for (int y = 0; y < result->height; y++) {
    for (int x = 0; x < result->width; x++) {
      tmp = *FLATMAT_AT(&host_similarly, x, y, 0) == max;
      *FLATMAT_AT(&result->map, x, y, 0) = tmp;
      result->cnt += tmp;
    }
  }

  // Cleanup
  flatmat_free_cuda(&device_src);
  flatmat_free_cuda(&device_tmpl);
  flatmat_free_cuda(&device_similarly);
  flatmat_free_cuda(&device_result);
  flatmat_free(&host_similarly);
}