extern "C" {
#include "benchmark.h"
}
#include "cudautil.h"
#include <stdio.h>

static hipEvent_t start, stop;

// [TODO] Benchmark using CUDA API will return invalid handle

/**
 * Begin time measurement
 */
void benchmark_gpu_begin() {

  GPU_ERRCHK(hipEventCreate(&start));
  GPU_ERRCHK(hipEventCreate(&stop));

  GPU_ERRCHK(hipEventRecord(start));
}

/**
 * End time measurement
 */
double benchmark_gpu_end() {
  float elapsed;
  hipEventRecord(stop);
  GPU_ERRCHK(hipEventSynchronize(stop));
  GPU_ERRCHK(hipEventElapsedTime(&elapsed, start, stop));
  elapsed /= 1.0E3;

  return (double)elapsed;
}