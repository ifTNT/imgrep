extern "C" {
#include "benchmark.h"
}

static hipEvent_t start, stop;

/**
 * Begin time measurement
 */
void benchmark_gpu_begin() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
}

/**
 * End time measurement
 */
double benchmark_gpu_end() {
  float elapsed;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1.0E3;

  return (double)elapsed;
}