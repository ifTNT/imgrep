#include "benchmark.hpp"

static hipEvent_t start, stop;

/**
 * Begin time measurement
 */
extern "C" void benchmark_gpu_begin() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
}

/**
 * End time measurement
 */
extern "C" double benchmark_gpu_end() {
  float elapsed;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  elapsed /= 1.0E3;

  return (double)elapsed;
}